#include "hip/hip_runtime.h"
#include <jni.h>
#include <hip/hip_runtime.h>

__device__ float distance(float2 a, float2 b)
{
    return hypotf(b.x - a.x, b.y - a.y);
}

__device__ float2 lerp(float2 a, float2 b, float t)
{
    return make_float2(fmaf(t, b.x - a.x, a.x), fmaf(t, b.y - a.y, a.y));
}

__global__ void castRaysKernel(
    int width,
    int maxSteps,
    float epsilon,
    float2 playerPos,
    float playerHeading,
    float2 leftMostRayDirection,
    float2 rightMostRayDirection,
    float *wallHeights,
    int *colors,
    int *map,
    int mapWidth,
    int mapHeight)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= width)
        return;

    float t = x / (float)width;
    float2 rayDir = lerp(leftMostRayDirection, rightMostRayDirection, t);
    float2 rayPos = playerPos;

    for (int step = 0; step < maxSteps; ++step)
    {
        int mapX = (int)floorf(rayPos.x);
        int mapY = (int)floorf(rayPos.y);

        if (mapX >= 0 && mapX < mapWidth && mapY >= 0 && mapY < mapHeight)
        {
            int color = map[mapY * mapWidth + mapX];
            if (color != 0)  // TODO: NULL instead of 0
            {
                float dist = distance(rayPos, playerPos);
                float adjustedDist = dist * cosf(rayDir.x - playerHeading);
                wallHeights[x] = 1.0f / fmaxf(1.0f, adjustedDist);
                colors[x] = color;
                return;
            }
        }

        // Step the ray
        float targetX = (rayDir.x > 0) ? floorf(rayPos.x + 1) : ceilf(rayPos.x - 1);
        float targetY = (rayDir.y > 0) ? floorf(rayPos.y + 1) : ceilf(rayPos.y - 1);
        float stepSize = fminf((targetX - rayPos.x) / rayDir.x, (targetY - rayPos.y) / rayDir.y);
        rayPos.x += rayDir.x * stepSize;
        rayPos.y += rayDir.y * stepSize;
    }
}

extern "C" JNIEXPORT void JNICALL Java_org_spi3lot_rendering_RaycastGpu_castCudaRays(
    JNIEnv *env,
    jobject obj,
    jobjectArray map,
    jint width,
    jint maxSteps,
    jfloat epsilon,
    jfloat playerX,
    jfloat playerY,
    jfloat playerHeading,
    jfloat leftMostRayDirectionX,
    jfloat leftMostRayDirectionY,
    jfloat rightMostRayDirectionX,
    jfloat rightMostRayDirectionY,
    jfloatArray wallHeightsArray,
    jintArray colorsArray)
{
    // Convert Java arrays to native arrays
    jfloat *wallHeights = env->GetFloatArrayElements(wallHeightsArray, 0);
    jint *colors = env->GetIntArrayElements(colorsArray, 0);

    // Get map data from Java object
    int mapHeight = env->GetArrayLength(map);
    int mapWidth = env->GetArrayLength((jintArray)env->GetObjectArrayElement(map, 0));

    // Allocate and copy map data to device
    int *h_map = new int[mapWidth * mapHeight];
    for (int i = 0; i < mapHeight; ++i)
    {
        jintArray row = (jintArray)env->GetObjectArrayElement(map, i);
        jint *rowData = env->GetIntArrayElements(row, 0);
        memcpy(h_map + i * mapWidth, rowData, mapWidth * sizeof(int));
        env->ReleaseIntArrayElements(row, rowData, 0);
    }

    int *d_map;
    hipMalloc(&d_map, mapWidth * mapHeight * sizeof(int));
    hipMemcpy(d_map, h_map, mapWidth * mapHeight * sizeof(int), hipMemcpyHostToDevice);
    delete[] h_map;

    // Allocate device memory for results
    float *d_wallHeights;
    int *d_colors;
    hipMalloc(&d_wallHeights, width * sizeof(float));
    hipMalloc(&d_colors, width * sizeof(int));

    // Define player position and ray directions
    float2 playerPos = make_float2(playerX, playerY);
    float2 leftMostRay = make_float2(leftMostRayDirectionX, leftMostRayDirectionY);
    float2 rightMostRay = make_float2(rightMostRayDirectionX, rightMostRayDirectionY);

    // Launch the kernel
    int blockSize = 256;
    int numBlocks = (width + blockSize - 1) / blockSize;

    castRaysKernel<<<numBlocks, blockSize>>>(
        width,
        maxSteps,
        epsilon,
        playerPos,
        playerHeading,
        leftMostRay,
        rightMostRay,
        d_wallHeights,
        d_colors,
        d_map,
        mapWidth,
        mapHeight);

    // Copy results back to host
    hipMemcpy(wallHeights, d_wallHeights, width * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(colors, d_colors, width * sizeof(int), hipMemcpyDeviceToHost);

    // Release device memory
    hipFree(d_wallHeights);
    hipFree(d_colors);
    hipFree(d_map);

    // Release Java arrays
    env->ReleaseFloatArrayElements(wallHeightsArray, wallHeights, 0);
    env->ReleaseIntArrayElements(colorsArray, colors, 0);
}