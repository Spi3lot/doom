#include "hip/hip_runtime.h"
#include <jni.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float distance(float2 a, float2 b)
{
    return hypotf(b.x - a.x, b.y - a.y);
}

__device__ float2 lerp(float2 a, float2 b, float t)
{
    return make_float2(fmaf(t, b.x - a.x, a.x), fmaf(t, b.y - a.y, a.y));
}

__global__ void castRaysKernel(
    int *map,
    int mapWidth,
    int mapHeight,
    float worldScale,
    int windowWidth,
    int windowHeight,
    int maxSteps,
    float epsilon,
    float2 playerPos,
    float playerHeading,
    float2 leftMostRayDirection,
    float2 rightMostRayDirection,
    int *wallHeights,
    int *colors)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= windowWidth)
        return;

    float t = x / (float)windowWidth;
    float2 rayPos = playerPos;
    float2 rayDir = lerp(leftMostRayDirection, rightMostRayDirection, t);
    float rayHeading = atan2f(rayDir.y, rayDir.x);

    for (int step = 0; step < maxSteps; ++step)
    {
        int mapX = (int)floorf(rayPos.x);
        int mapY = (int)floorf(rayPos.y);

        if (mapX >= 0 && mapX < mapWidth && mapY >= 0 && mapY < mapHeight)
        {
            int color = map[mapY * mapWidth + mapX];

            if (color != 0) // TODO: NULL instead of 0
            {
                float dist = worldScale * distance(rayPos, playerPos);
                float adjustedDist = dist * cosf(rayHeading - playerHeading);
                wallHeights[x] = (int) (windowHeight / fmaxf(1.0f, adjustedDist));
                colors[x] = color;
                return;
            }
        }

        // Step the ray
        float targetX = (rayDir.x > 0) ? floorf(rayPos.x + 1) : ceilf(rayPos.x - 1);
        float targetY = (rayDir.y > 0) ? floorf(rayPos.y + 1) : ceilf(rayPos.y - 1);
        float stepSize = fminf((targetX - rayPos.x) / rayDir.x, (targetY - rayPos.y) / rayDir.y);
        rayPos.x += rayDir.x * (stepSize + epsilon);
        rayPos.y += rayDir.y * (stepSize + epsilon);
    }
}

extern "C" JNIEXPORT void JNICALL Java_org_spi3lot_rendering_RaycastGpu_castCudaRays(
    JNIEnv *env,
    jobject obj,
    jobjectArray map,
    jfloat worldScale,
    jint windowWidth,
    jint windowHeight,
    jint maxSteps,
    jfloat epsilon,
    jfloat playerX,
    jfloat playerY,
    jfloat playerHeading,
    jfloat leftMostRayDirectionX,
    jfloat leftMostRayDirectionY,
    jfloat rightMostRayDirectionX,
    jfloat rightMostRayDirectionY,
    jintArray wallHeightsArray,
    jintArray colorsArray)
{
    // Convert Java arrays to native arrays
    jint *wallHeights = env->GetIntArrayElements(wallHeightsArray, 0);
    jint *colors = env->GetIntArrayElements(colorsArray, 0);

    // Get map data from Java object
    int mapHeight = env->GetArrayLength(map);
    int mapWidth = env->GetArrayLength((jintArray)env->GetObjectArrayElement(map, 0));

    // Allocate and copy map data to device
    int *h_map = new int[mapWidth * mapHeight];
    for (int i = 0; i < mapHeight; ++i)
    {
        jintArray row = (jintArray)env->GetObjectArrayElement(map, i);
        int *rowData = env->GetIntArrayElements(row, 0);
        memcpy(h_map + i * mapWidth, rowData, mapWidth * sizeof(int));
        env->ReleaseIntArrayElements(row, rowData, 0);
    }

    int *d_map;
    hipMalloc(&d_map, mapWidth * mapHeight * sizeof(int));
    hipMemcpy(d_map, h_map, mapWidth * mapHeight * sizeof(int), hipMemcpyHostToDevice);
    delete[] h_map;

    // Allocate device memory for results
    int *d_wallHeights;
    int *d_colors;
    hipMalloc(&d_wallHeights, windowWidth * sizeof(float));
    hipMalloc(&d_colors, windowWidth * sizeof(int));

    // Define player position and ray directions
    float2 playerPos = make_float2(playerX, playerY);
    float2 leftMostRay = make_float2(leftMostRayDirectionX, leftMostRayDirectionY);
    float2 rightMostRay = make_float2(rightMostRayDirectionX, rightMostRayDirectionY);

    // Launch the kernel
    int blockSize = 256;
    int numBlocks = (windowWidth + blockSize - 1) / blockSize;

    castRaysKernel<<<numBlocks, blockSize>>>(
        d_map,
        mapWidth,
        mapHeight,
        worldScale,
        windowWidth,
        windowHeight,
        maxSteps,
        epsilon,
        playerPos,
        playerHeading,
        leftMostRay,
        rightMostRay,
        d_wallHeights,
        d_colors);

    // Copy results back to host
    hipMemcpy(wallHeights, d_wallHeights, windowWidth * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(colors, d_colors, windowWidth * sizeof(int), hipMemcpyDeviceToHost);

    // Release device memory
    hipFree(d_wallHeights);
    hipFree(d_colors);
    hipFree(d_map);

    // Release Java arrays
    env->ReleaseIntArrayElements(wallHeightsArray, wallHeights, 0);
    env->ReleaseIntArrayElements(colorsArray, colors, 0);
}